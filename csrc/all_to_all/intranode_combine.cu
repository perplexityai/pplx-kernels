#include "hip/hip_runtime.h"
#include "all_to_all/intranode.cuh"
#include "core/atomic.cuh"
#include "core/device_utils.h"
#include "core/utils.h"
#include "intranode.h"

#include <cassert>

#include <hip/hip_cooperative_groups.h>
#include <nvtx3/nvToolsExt.h>

using namespace pplx;

namespace {

template <typename T, typename U, size_t NUM_WARPS, bool DO_SEND, bool DO_RECV>
__global__ __launch_bounds__(NUM_WARPS * 32, 1) void combineKernel(
    U *outTokens,
    size_t outTokensStrideElem,
    uint32_t *indices,
    size_t indicesStrideElem,
    size_t indicesStrideRow,
    float *weights,
    size_t weightsStrideElem,
    size_t weightsStrideRow,
    T *expertX,
    size_t expertXStrideElem,
    size_t expertXStrideRow,
    size_t expertsPerToken,
    size_t maxNumTokens,
    size_t numExperts,
    unsigned rank,
    unsigned worldSize,
    unsigned dpSize,
    size_t hiddenDim,
    unsigned *boundM,
    unsigned m,
    std::byte **sendBuffersPtr,
    std::byte **recvBuffersPtr,
    const uint32_t *sourceExpert,
    const uint32_t *sourceIndex,
    const uint32_t *sourceOffset,
    const uint32_t *sourceRank,
    uint32_t &globalTokenIndex
) {
  const unsigned numLocalExperts = numExperts / worldSize;
  const size_t tokenDim = hiddenDim * sizeof(T);
  const size_t tokenStride = device::round_up<size_t>(tokenDim, sizeof(int4));
  constexpr unsigned WARP_SIZE = 32;
  uint32_t warpId = threadIdx.x / WARP_SIZE;
  const unsigned laneId = threadIdx.x % WARP_SIZE;

  BufferWrapper remoteBuffer(recvBuffersPtr, numLocalExperts, worldSize, maxNumTokens, tokenStride);
  BufferWrapper localBuffer(sendBuffersPtr, numLocalExperts, worldSize, maxNumTokens, tokenStride);

  if (DO_SEND) {
    size_t numSendTokens = globalTokenIndex;

    for (unsigned i = blockIdx.x * blockDim.x + threadIdx.x; i < worldSize;
         i += blockDim.x * gridDim.x) {
      while (ld_flag_volatile(&localBuffer.getDispatchSyncPtr(i)) != 0)
        ;
      st_flag_volatile(&remoteBuffer.getCombineSyncPtr(i), 1);
    }

    // Dispatch the tokens from the expert to the DP groups.
    for (uint32_t token = blockIdx.x; token < numSendTokens; token += gridDim.x) {
      const uint32_t expert = __ldg(&sourceExpert[token]);
      const uint32_t index = __ldg(&sourceIndex[token]);
      const uint32_t offset = __ldg(&sourceOffset[token]);
      const uint32_t rank = __ldg(&sourceRank[token]);

      const uint32_t dstLocalExpert = expert % numLocalExperts;

      const T *source = expertX + expert * expertXStrideRow + offset * expertXStrideElem;
      const unsigned n = tokenDim / sizeof(float4);

      auto copy = [&](unsigned rank, unsigned start, unsigned step) {
        std::byte *buffer = remoteBuffer.getTokenPtr(rank, dstLocalExpert, index);
        float4 *srcPtr = (float4 *)source;
        float4 *dstPtr = (float4 *)buffer;

        srcPtr += start;
        dstPtr += start;

#pragma unroll(4)
        for (unsigned j = start; j < n; j += step) {
          *dstPtr = *srcPtr;
          dstPtr += step;
          srcPtr += step;
        }
      };

      if (dpSize == 1) {
        copy(rank, threadIdx.x, blockDim.x);
      } else {
        for (unsigned i = warpId; i < dpSize; i += NUM_WARPS) {
          copy((rank / dpSize) * dpSize + i, laneId, WARP_SIZE);
        }
      }
    }

    cooperative_groups::this_grid().sync();

    for (unsigned i = blockIdx.x * blockDim.x + threadIdx.x; i < worldSize;
         i += blockDim.x * gridDim.x) {
      st_flag_release(&remoteBuffer.getCountPtr(i, 0), 1);
    }
  }

  // Synchronize the grid to ensure that tokens routed within the rank are
  // correctly transported from one block to another.
  if (DO_RECV) {
    for (unsigned i = blockIdx.x * blockDim.x + threadIdx.x; i < worldSize;
         i += blockDim.x * gridDim.x) {
      while (ld_flag_acquire(&localBuffer.getCountPtr(i, 0)) == 0)
        ;
      st_flag_volatile(&localBuffer.getCountPtr(i, 0), 0);
    }

    cooperative_groups::this_grid().sync();
    globalTokenIndex = 0;

    // Compute the weighed sum of the input tokens.
    const size_t numRecvTokens = boundM ? __ldg(boundM) : m;
    for (unsigned i = blockIdx.x; i < numRecvTokens; i += gridDim.x) {
      U *dstPtr = outTokens + i * outTokensStrideElem;
      constexpr unsigned VEC_SIZE = 8;
      for (unsigned j = threadIdx.x * VEC_SIZE; j < hiddenDim; j += blockDim.x * VEC_SIZE) {
        float sum[VEC_SIZE];

#pragma unroll
        for (unsigned l = 0; l < VEC_SIZE; ++l) {
          sum[l] = 0.0f;
        }

        for (unsigned k = 0; k < expertsPerToken; ++k) {
          const uint32_t expert = indices[i * expertsPerToken + k];
          const uint32_t dstRank = expert / numLocalExperts;
          const uint32_t dstLocalExpert = expert % numLocalExperts;
          const float weight = weights[i * weightsStrideRow + k];

#pragma unroll
          for (unsigned l = 0; l < VEC_SIZE; ++l) {
            std::byte *buffer = localBuffer.getTokenPtr(dstRank, dstLocalExpert, i);
            sum[l] += weight * (float)((T *)buffer)[j + l];
          }
        }

#pragma unroll
        for (unsigned l = 0; l < VEC_SIZE; ++l) {
          dstPtr[j + l] = sum[l];
        }
      }
    }

    for (unsigned i = blockIdx.x * blockDim.x + threadIdx.x; i < worldSize;
         i += blockDim.x * gridDim.x) {
      st_flag_volatile(&remoteBuffer.getCombineSyncPtr(i), 0);
    }
  }
}
} // namespace

template <typename T, typename U>
void AllToAllIntraNode::combine(
    const Strided1D<U> &outTokens,
    const Strided2D<uint32_t> &indices,
    const Strided2D<float> &weights,
    const Strided2D<T> &expertX,
    unsigned m,
    const unsigned *boundM,
    SplitMode splitMode,
    hipStream_t stream
) {
  constexpr size_t NUM_WARPS = 32;

  const size_t numLocalExperts = numExperts / worldSize;
  const size_t numDPGroups = worldSize / dpSize;
  const size_t batchNumTokens = numLocalExperts * numDPGroups * maxNumTokens;
  const size_t numBlocks = std::min(132ul, batchNumTokens);

  assert(hiddenDimBytes % 16 == 0);

  dim3 dimGrid(numBlocks, 1, 1);
  dim3 dimBlock(NUM_WARPS * 32, 1, 1);

  void *args[] = {
      const_cast<U **>(&outTokens.data),
      const_cast<size_t *>(&outTokens.strideElem),
      const_cast<uint32_t **>(&indices.data),
      const_cast<size_t *>(&indices.strideElem),
      const_cast<size_t *>(&indices.strideRow),
      const_cast<float **>(&weights.data),
      const_cast<size_t *>(&weights.strideElem),
      const_cast<size_t *>(&weights.strideRow),
      const_cast<T **>(&expertX.data),
      const_cast<size_t *>(&expertX.strideElem),
      const_cast<size_t *>(&expertX.strideRow),
      const_cast<size_t *>(&expertsPerToken),
      const_cast<size_t *>(&maxNumTokens),
      const_cast<size_t *>(&numExperts),
      const_cast<unsigned *>(&rank),
      const_cast<unsigned *>(&worldSize),
      const_cast<unsigned *>(&dpSize),
      const_cast<size_t *>(&hiddenDim),
      const_cast<unsigned **>(&boundM),
      &m,
      &sendBuffersPtr,
      &recvBuffersPtr,
      &sourceExpert,
      &sourceIndex,
      &sourceOffset,
      &sourceRank,
      &tokenIndex,
  };

  nvtxRangePush("combine");
  switch (splitMode) {
  case SplitMode::SEND:
    CUDACHECK(hipLaunchCooperativeKernel(
        (void *)&combineKernel<T, U, NUM_WARPS, true, false>, dimGrid, dimBlock, args, 0, stream
    ));
    break;
  case SplitMode::RECV:
    CUDACHECK(hipLaunchCooperativeKernel(
        (void *)&combineKernel<T, U, NUM_WARPS, false, true>, dimGrid, dimBlock, args, 0, stream
    ));
    break;
  case SplitMode::NONE:
    CUDACHECK(hipLaunchCooperativeKernel(
        (void *)&combineKernel<T, U, NUM_WARPS, true, true>, dimGrid, dimBlock, args, 0, stream
    ));
    break;
  default:
    ROSE_UNREACHABLE("invalid split mode");
  }
  nvtxRangePop();
}

#define INSTANTIATE_COMBINE(T, U)                                                                  \
  template void AllToAllIntraNode::combine<T, U>(                                                  \
      const Strided1D<U> &outTokens,                                                               \
      const Strided2D<uint32_t> &indices,                                                          \
      const Strided2D<float> &weights,                                                             \
      const Strided2D<T> &expertX,                                                                 \
      unsigned m,                                                                                  \
      const unsigned *boundM,                                                                      \
      SplitMode splitMode,                                                                         \
      hipStream_t stream                                                                          \
  );

INSTANTIATE_COMBINE(float, hip_bfloat16)
INSTANTIATE_COMBINE(float, half)
INSTANTIATE_COMBINE(half, hip_bfloat16)
INSTANTIATE_COMBINE(half, half)
INSTANTIATE_COMBINE(hip_bfloat16, hip_bfloat16)
INSTANTIATE_COMBINE(hip_bfloat16, half)
