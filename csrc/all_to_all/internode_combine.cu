#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <nvshmem.h>
#include <nvtx3/nvToolsExt.h>

#include "all_to_all/internode.h"
#include "core/nvshmem_utils.h"
#include "core/utils.h"

using namespace pplx;

template <typename T, typename U, size_t NUM_WARPS, bool DO_SEND, bool DO_RECV>
__global__ __launch_bounds__(NUM_WARPS * 32, 1) void combineKernel(
    U *outTokens,
    size_t outTokensStrideElem,
    uint32_t *indices,
    size_t indicesStrideElem,
    size_t indicesStrideRow,
    float *weights,
    size_t weightsStrideElem,
    size_t weightsStrideRow,
    T *expertX,
    size_t expertXStrideElem,
    size_t expertXStrideRow,
    size_t expertsPerToken,
    size_t maxNumTokens,
    size_t numExperts,
    unsigned rank,
    unsigned worldSize,
    unsigned dpSize,
    size_t hiddenDim,
    unsigned *boundM,
    unsigned m,
    uint32_t *sourceExpert,
    const uint32_t *sourceIndex,
    const uint32_t *sourceOffset,
    const uint32_t *sourceGroup,
    uint64_t *combineSignalBuffer,
    uint64_t *combineSyncBuffer,
    std::byte *xBufferIn,
    std::byte *xBufferOut
) {
  const unsigned numLocalExperts = numExperts / worldSize;
  const unsigned numDPGroups = worldSize / dpSize;
  const unsigned maxBatchTokens = numLocalExperts * numDPGroups * maxNumTokens;
  const size_t stride = hiddenDim * sizeof(T);
  uint32_t warpId = threadIdx.x / 32;
  const unsigned laneId = threadIdx.x % 32;
  const unsigned numWarps = blockDim.x / 32;

  if (DO_SEND) {
    for (unsigned i = blockIdx.x * numWarps + warpId; i < worldSize; i += gridDim.x * numWarps) {
      if (laneId == 0) {
        nvshmemx_signal_op(&combineSyncBuffer[rank], 1, NVSHMEM_SIGNAL_SET, i);
      }
    }

    // Dispatch the tokens from the expert to the DP groups.
    for (uint32_t token = blockIdx.x; token < maxBatchTokens; token += gridDim.x) {
      const uint32_t expertPlusOne = sourceExpert[token];
      __syncthreads();
      if (expertPlusOne == 0) {
        break;
      }
      sourceExpert[token] = 0;

      const uint32_t source = __ldg(&sourceIndex[token]);
      const uint32_t offset = __ldg(&sourceOffset[token]);
      const uint32_t dp = __ldg(&sourceGroup[token]);
      const uint32_t expert = expertPlusOne - 1;

      // Copy the token to shared memory for send.
      const int4 *expertXTokenPtr =
          (int4 *)(expertX + expert * expertXStrideRow + offset * expertXStrideElem);
      int4 *xTokenPtr = (int4 *)(xBufferIn + token * stride);
      for (unsigned j = threadIdx.x; j * sizeof(int4) < stride; j += blockDim.x) {
        xTokenPtr[j] = expertXTokenPtr[j];
      }
      __syncthreads();

      const uint32_t dstExpert = rank * numLocalExperts + expert;

      for (unsigned i = warpId; i < dpSize; i += numWarps) {
        const int dstRank = dp * dpSize + i;
        const unsigned index = dstExpert * maxNumTokens + source;
        std::byte *dstPtr = xBufferOut + index * stride;
        nvshmemx_putmem_signal_nbi_warp(
            dstPtr, xTokenPtr, stride, &combineSignalBuffer[source], 1, NVSHMEM_SIGNAL_ADD, dstRank
        );
      }
    }
  }

  // Synchronize the grid to ensure that tokens routed within the rank are
  // correctly transported from one block to another.
  if (DO_RECV) {
    if (DO_SEND) {
      cg::this_grid().sync();
    }

    // Compute the weighed sum of the input tokens.
    const size_t localNumTokens = boundM ? __ldg(boundM) : m;
    for (unsigned i = blockIdx.x; i < localNumTokens; i += gridDim.x) {
      nvshmem_uint64_wait_until(&combineSignalBuffer[i], NVSHMEM_CMP_EQ, expertsPerToken);
      __syncthreads();
      combineSignalBuffer[i] = 0;

      U *dstPtr = outTokens + i * outTokensStrideElem;
      constexpr unsigned VEC_SIZE = 8;
      for (unsigned j = threadIdx.x * VEC_SIZE; j < hiddenDim; j += blockDim.x * VEC_SIZE) {
        float sum[VEC_SIZE];

#pragma unroll
        for (unsigned l = 0; l < VEC_SIZE; ++l) {
          sum[l] = 0.0f;
        }

        for (unsigned k = 0; k < expertsPerToken; ++k) {
          const uint32_t expert = indices[i * expertsPerToken + k];
          const float weight = weights[i * weightsStrideRow + k];

#pragma unroll
          for (unsigned l = 0; l < VEC_SIZE; ++l) {
            std::byte *xDstPtr = xBufferOut + (expert * maxNumTokens + i) * stride;
            sum[l] += weight * (float)((T *)xDstPtr)[j + l];
          }
        }

#pragma unroll
        for (unsigned l = 0; l < VEC_SIZE; ++l) {
          dstPtr[j + l] = sum[l];
        }
      }
    }

    for (unsigned i = blockIdx.x * blockDim.x + threadIdx.x; i < worldSize;
         i += gridDim.x * blockDim.x) {
      nvshmem_uint64_wait_until(&combineSyncBuffer[i], NVSHMEM_CMP_EQ, 1);
      combineSyncBuffer[i] = 0;
    }
  }
}

template <typename T, typename U>
void AllToAllInterNode::combine(
    const Strided1D<U> &outTokens,
    const Strided2D<uint32_t> &indices,
    const Strided2D<float> &weights,
    const Strided2D<T> &expertX,
    unsigned m,
    const unsigned *boundM,
    SplitMode splitMode,
    hipStream_t stream
) {

  constexpr size_t NUM_WARPS = 8;

  const size_t numLocalExperts = numExperts / worldSize;
  const size_t numDPGroups = worldSize / dpSize;
  const size_t batchNumTokens = numLocalExperts * numDPGroups * maxNumTokens;
  const size_t numBlocks = std::min(132ul, batchNumTokens);

  assert(hiddenDimBytes % 16 == 0);

  dim3 dimGrid(numBlocks, 1, 1);
  dim3 dimBlock(NUM_WARPS * 32, 1, 1);

  void *args[] = {
      const_cast<U **>(&outTokens.data),
      const_cast<size_t *>(&outTokens.strideElem),
      const_cast<uint32_t **>(&indices.data),
      const_cast<size_t *>(&indices.strideElem),
      const_cast<size_t *>(&indices.strideRow),
      const_cast<float **>(&weights.data),
      const_cast<size_t *>(&weights.strideElem),
      const_cast<size_t *>(&weights.strideRow),
      const_cast<T **>(&expertX.data),
      const_cast<size_t *>(&expertX.strideElem),
      const_cast<size_t *>(&expertX.strideRow),
      const_cast<size_t *>(&expertsPerToken),
      const_cast<size_t *>(&maxNumTokens),
      const_cast<size_t *>(&numExperts),
      const_cast<unsigned *>(&rank),
      const_cast<unsigned *>(&worldSize),
      const_cast<unsigned *>(&dpSize),
      const_cast<size_t *>(&hiddenDim),
      const_cast<unsigned **>(&boundM),
      &m,
      &sourceExpert,
      &sourceIndex,
      &sourceOffset,
      &sourceGroup,
      &combineSignalBuffer,
      &combineSyncBuffer,
      &xCombineIn,
      &xCombineOut};

  nvtxRangePush("combine");
  switch (splitMode) {
  case SplitMode::SEND:
    CUDACHECK(hipLaunchCooperativeKernel(
        (void *)&combineKernel<T, U, NUM_WARPS, true, false>, dimGrid, dimBlock, args, 0, stream
    ));
    break;
  case SplitMode::RECV:
    CUDACHECK(hipLaunchCooperativeKernel(
        (void *)&combineKernel<T, U, NUM_WARPS, false, true>, dimGrid, dimBlock, args, 0, stream
    ));
    break;
  case SplitMode::NONE:
    CUDACHECK(hipLaunchCooperativeKernel(
        (void *)&combineKernel<T, U, NUM_WARPS, true, true>, dimGrid, dimBlock, args, 0, stream
    ));
    break;
  default:
    ROSE_UNREACHABLE("invalid split mode");
  }
  nvtxRangePop();
}

#define INSTANTIATE_COMBINE(T, U)                                                                  \
  template void AllToAllInterNode::combine<T, U>(                                                  \
      const Strided1D<U> &outTokens,                                                               \
      const Strided2D<uint32_t> &indices,                                                          \
      const Strided2D<float> &weights,                                                             \
      const Strided2D<T> &expertX,                                                                 \
      unsigned m,                                                                                  \
      const unsigned *boundM,                                                                      \
      SplitMode splitMode,                                                                         \
      hipStream_t stream                                                                          \
  );

INSTANTIATE_COMBINE(float, hip_bfloat16)
INSTANTIATE_COMBINE(half, hip_bfloat16)
INSTANTIATE_COMBINE(hip_bfloat16, hip_bfloat16)
INSTANTIATE_COMBINE(float, half)
INSTANTIATE_COMBINE(half, half)
INSTANTIATE_COMBINE(hip_bfloat16, half)
